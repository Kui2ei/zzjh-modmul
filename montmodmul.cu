
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#ifndef __CUDA_ARCH__
#define __CUDA_ARCH__
#endif

# include <cstddef>
# include <cstdint>

# define inline __device__ __forceinline__
# ifdef __GNUC__
#  define asm __asm__ __volatile__
# else
#  define asm asm volatile
# endif

// 0x0e0a77c19a07df2f666ea36f7879462c0a78eb28f5c70b3dd35d438dc58f0d9d
// 0x0e0a77c19a07df2f666ea36f7879462c0a78eb28f5c70b3dd35d438dc58f0d9d
namespace device {
#define TO_CUDA_T(limb64) (uint32_t)(limb64), (uint32_t)(limb64>>32)
    static __device__ __constant__ __align__(16) const uint32_t ALT_BN128_P[8] = {
        TO_CUDA_T(0x3c208c16d87cfd47), TO_CUDA_T(0x97816a916871ca8d),
        TO_CUDA_T(0xb85045b68181585d), TO_CUDA_T(0x30644e72e131a029)
    };
    static __device__ __constant__ __align__(16) const uint32_t ALT_BN128_RR[8] = { /* (1<<512)%P */
        TO_CUDA_T(0xf32cfc5b538afa89), TO_CUDA_T(0xb5e71911d44501fb),
        TO_CUDA_T(0x47ab1eff0a417ff6), TO_CUDA_T(0x06d89f71cab8351f),
    };
    static __device__ __constant__ __align__(16) const uint32_t ALT_BN128_one[8] = { /* (1<<256)%P */
        TO_CUDA_T(0xd35d438dc58f0d9d), TO_CUDA_T(0x0a78eb28f5c70b3d),
        TO_CUDA_T(0x666ea36f7879462c), TO_CUDA_T(0x0e0a77c19a07df2f)
    };
    static __device__ __constant__ __align__(16) const uint32_t ALT_BN128_Px4[8] = { /* left-aligned value of the modulus */
        TO_CUDA_T(0xf082305b61f3f51c), TO_CUDA_T(0x5e05aa45a1c72a34),
        TO_CUDA_T(0xe14116da06056176), TO_CUDA_T(0xc19139cb84c680a6)
    };
    //n`(rr_1-nn`=1) 0xd76b780d00bad4acbbc3ac183dcfc3f073dd0cc126dc4ece43b5e10de0f8076
    static __device__ __constant__ const uint32_t ALT_BN128_M0 = 0xe4866389;


    static __device__ __constant__ __align__(16) const uint32_t MNT4753_Fr_P[24] = { /* left-aligned value of the modulus */
        TO_CUDA_T(0xd90776e240000001), TO_CUDA_T(0x4ea099170fa13a4f),
        TO_CUDA_T(0xd6c381bc3f005797), TO_CUDA_T(0xb9dff97634993aa4),
        TO_CUDA_T(0x3eebca9429212636), TO_CUDA_T(0xb26c5c28c859a99b),
        TO_CUDA_T(0x99d124d9a15af79d), TO_CUDA_T(0x07fdb925e8a0ed8d),
        TO_CUDA_T(0x5eb7e8f96c97d873), TO_CUDA_T(0xb7f997505b8fafed),
        TO_CUDA_T(0x10229022eee2cdad), TO_CUDA_T(0x0001c4c62d92c411)
    };

    
    static __device__ __constant__ const uint32_t MNT4753_Fr_M0 = 0x3fffffff;


    static __device__ __constant__ __align__(16) const uint32_t BLS12_377_P[12] = {
        TO_CUDA_T(0x8508c00000000001), TO_CUDA_T(0x170b5d4430000000),
        TO_CUDA_T(0x1ef3622fba094800), TO_CUDA_T(0x1a22d9f300f5138f),
        TO_CUDA_T(0xc63b05c06ca1493b), TO_CUDA_T(0x01ae3a4617c510ea)
    };

    
    static __device__ __constant__ /*const*/ uint32_t BLS12_377_M0 = 0xffffffff;

    
 static __device__ __constant__ __align__(16) const uint32_t Curve1024_P0[32] = { /* left-aligned value of the modulus */
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xfffffffffffffffe), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff)
    };



    static __device__ __constant__ /*const*/ uint32_t Curve1024_M0 = 0x1;


    static __device__ __constant__ __align__(16) const uint32_t Curve896_P0[28] = { /* left-aligned value of the modulus */
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xfffffffffffffffe),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff),
        TO_CUDA_T(0xffffffffffffffff), TO_CUDA_T(0xffffffffffffffff)
    };



    static __device__ __constant__ /*const*/ uint32_t Curve896_M0 = 0x1;

}


template<const size_t N, const uint32_t MOD[(N+31)/32], const uint32_t& M0,
         const uint32_t RR[(N+31)/32], const uint32_t ONE[(N+31)/32],
         const uint32_t MODx[(N+31)/32] = MOD>
class __align__(((N+63)/64)&1 ? 8 : 16) mont_t {
        
    public:
        static const size_t nbits = N;
        static constexpr size_t __device__ bit_length() { return N; }
        static const uint32_t degree = 1;
        using mem_t = mont_t;
        static const size_t n = (N+31)/32;
        
    public:
        uint32_t even[n];

    private:
        static inline void mul_n(uint32_t* acc, const uint32_t* a, uint32_t bi,
        size_t n_=n)
        {
            for (size_t j = 0; j < n_; j += 2)
                asm("mul.lo.u32 %0, %2, %3; mul.hi.u32 %1, %2, %3;"
                    : "=r"(acc[j]), "=r"(acc[j+1])
                    : "r"(a[j]), "r"(bi));
        }

        static inline void cmad_n(uint32_t* acc, const uint32_t* a, uint32_t bi,
                                size_t n_=n)
        {
            asm("mad.lo.cc.u32 %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
                : "+r"(acc[0]), "+r"(acc[1])
                : "r"(a[0]), "r"(bi));
            for (size_t j = 2; j < n_; j += 2)
                asm("madc.lo.cc.u32 %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
                    : "+r"(acc[j]), "+r"(acc[j+1])
                    : "r"(a[j]), "r"(bi));
            // return carry flag
        }

        static inline void cadd_n(uint32_t* acc, const uint32_t* a, size_t n_=n)
        {
            asm("add.cc.u32 %0, %0, %1;" : "+r"(acc[0]) : "r"(a[0]));
            for (size_t i = 1; i < n_; i++)
                asm("addc.cc.u32 %0, %0, %1;" : "+r"(acc[i]) : "r"(a[i]));
            // return carry flag
        }

        class wide_t {
        private:
            union {
                uint32_t even[2*n];
                mont_t s[2];
            };

        public:
            inline uint32_t& operator[](size_t i)               { return even[i]; }
            inline const uint32_t& operator[](size_t i) const   { return even[i]; }
            inline operator mont_t()
            {
                s[0].mul_by_1();
                return s[0] + s[1];
            }
            inline void final_sub(uint32_t carry, uint32_t* tmp)
            {   s[1].final_sub(carry, tmp);   }

            inline wide_t() {}

        private:
            static inline void mad_row(uint32_t* odd, uint32_t* even,
                                    const uint32_t* a, uint32_t bi, size_t n_=n)
            {
                cmad_n(odd, a+1, bi, n_-2);
                asm("madc.lo.cc.u32 %0, %2, %3, 0; madc.hi.u32 %1, %2, %3, 0;"
                    : "=r"(odd[n_-2]), "=r"(odd[n_-1])
                    : "r"(a[n_-1]), "r"(bi));

                cmad_n(even, a, bi, n_);
                asm("addc.u32 %0, %0, 0;" : "+r"(odd[n_-1]));
            }

        public:
            inline wide_t(const mont_t& a, const mont_t& b)     //// |a|*|b|
            {
                size_t i = 0;
                uint32_t odd[2*n-2];

                mul_n(even, &a[0], b[0]);
                mul_n(odd,  &a[1], b[0]);
                ++i; mad_row(&even[i+1], &odd[i-1], &a[0], b[i]);

                #pragma unroll
                while (i < n-2) {
                    ++i; mad_row(&odd[i],    &even[i],  &a[0], b[i]);
                    ++i; mad_row(&even[i+1], &odd[i-1], &a[0], b[i]);
                }

                // merge |even| and |odd|
                cadd_n(&even[1], &odd[0], 2*n-2);
                asm("addc.u32 %0, %0, 0;" : "+r"(even[2*n-1]));
            }
        private:
            static inline void qad_row(uint32_t* odd, uint32_t* even,
                                    const uint32_t* a, uint32_t bi, size_t n)
            {
                cmad_n(odd, a, bi, n-2);
                asm("madc.lo.cc.u32 %0, %2, %3, 0; madc.hi.u32 %1, %2, %3, 0;"
                    : "=r"(odd[n-2]), "=r"(odd[n-1])
                    : "r"(a[n-2]), "r"(bi));

                cmad_n(even, a+1, bi, n-2);
                asm("addc.u32 %0, %0, 0;" : "+r"(odd[n-1]));
            }
        public:
            inline wide_t(const mont_t& a)                      //// |a|**2
            {
                size_t i = 0, j;
                uint32_t odd[2*n-2];

                // perform |a[i]|*|a[j]| for all j>i
                mul_n(even+2, &a[2], a[0], n-2);
                mul_n(odd,    &a[1], a[0], n);

                #pragma unroll
                while (i < n-4) {
                    ++i; mad_row(&even[2*i+2], &odd[2*i], &a[i+1], a[i], n-i-1);
                    ++i; qad_row(&odd[2*i], &even[2*i+2], &a[i+1], a[i], n-i);
                }

                asm("mul.lo.u32 %0, %2, %3; mul.hi.u32 %1, %2, %3;"
                    : "=r"(even[2*n-4]), "=r"(even[2*n-3])
                    : "r"(a[n-1]), "r"(a[n-3]));
                asm("mad.lo.cc.u32 %0, %2, %3, %0; madc.hi.cc.u32 %1, %2, %3, %1;"
                    : "+r"(odd[2*n-6]), "+r"(odd[2*n-5])
                    : "r"(a[n-2]), "r"(a[n-3]));
                asm("addc.u32 %0, %0, 0;" : "+r"(even[2*n-3]));

                asm("mul.lo.u32 %0, %2, %3; mul.hi.u32 %1, %2, %3;"
                    : "=r"(odd[2*n-4]), "=r"(odd[2*n-3])
                    : "r"(a[n-1]), "r"(a[n-2]));

                // merge |even[2:]| and |odd[1:]|
                cadd_n(&even[2], &odd[1], 2*n-4);
                asm("addc.u32 %0, %1, 0;" : "=r"(even[2*n-2]) : "r"(odd[2*n-3]));

                // double |even|
                even[0] = 0;
                asm("add.cc.u32 %0, %1, %1;" : "=r"(even[1]) : "r"(odd[0]));
                for (j = 2; j < 2*n-1; j++)
                    asm("addc.cc.u32 %0, %0, %0;" : "+r"(even[j]));
                asm("addc.u32 %0, 0, 0;" : "=r"(even[j]));

                // accumulate "diagonal" |a[i]|*|a[i]| product
                i = 0;
                asm("mad.lo.cc.u32 %0, %2, %2, %0; madc.hi.cc.u32 %1, %2, %2, %1;"
                    : "+r"(even[2*i]), "+r"(even[2*i+1])
                    : "r"(a[i]));
                for (++i; i < n; i++)
                    asm("madc.lo.cc.u32 %0, %2, %2, %0; madc.hi.cc.u32 %1, %2, %2, %1;"
                        : "+r"(even[2*i]), "+r"(even[2*i+1])
                        : "r"(a[i]));
            }
        };

    private:
        inline operator const uint32_t*() const             { return even;    }
        inline operator uint32_t*()                         { return even;    }

    public:
        inline uint32_t& operator[](size_t i)               { return even[i]; }
        inline const uint32_t& operator[](size_t i) const   { return even[i]; }
        inline size_t len() const                           { return n;       }

        inline mont_t() {}
        // write in 
        inline mont_t(const uint32_t *p)
        {
            for (size_t i = 0; i < n; i++)
                even[i] = p[i];
        }


    // write back
        inline void store(uint32_t *p) const
        {
            for (size_t i = 0; i < n; i++)
                p[i] = even[i];
        }

        inline mont_t& operator+=(const mont_t& b)
        {
            cadd_n(&even[0], &b[0]);
            final_subc();
            return *this;
        }
        friend inline mont_t operator+(mont_t a, const mont_t& b)
        {   return a += b;   }

        friend inline mont_t operator<<(mont_t a, unsigned l)
        {   return a <<= l;   }

        inline mont_t& operator>>=(unsigned r)
        {
            size_t i;
            uint32_t tmp[n+1];

            while (r--) {
                tmp[n] = 0 - (even[0]&1);
                for (i = 0; i < n; i++)
                    tmp[i] = MOD[i] & tmp[n];

                cadd_n(&tmp[0], &even[0]);
                if (N%32 == 0)
                    asm("addc.u32 %0, 0, 0;" : "=r"(tmp[n]));

                for (i = 0; i < n-1; i++)
                    asm("shf.r.wrap.b32 %0, %1, %2, 1;"
                        : "=r"(even[i]) : "r"(tmp[i]), "r"(tmp[i+1]));
                if (N%32 == 0)
                    asm("shf.r.wrap.b32 %0, %1, %2, 1;"
                        : "=r"(even[i]) : "r"(tmp[i]), "r"(tmp[i+1]));
                else
                    even[i] = tmp[i] >> 1;
            }

            return *this;
        }
        friend inline mont_t operator>>(mont_t a, unsigned r)
        {   return a >>= r;   }

        inline mont_t& operator-=(const mont_t& b)
        {
            size_t i;
            uint32_t tmp[n], borrow;

            asm("sub.cc.u32 %0, %0, %1;" : "+r"(even[0]) : "r"(b[0]));
            for (i = 1; i < n; i++)
                asm("subc.cc.u32 %0, %0, %1;" : "+r"(even[i]) : "r"(b[i]));
            asm("subc.u32 %0, 0, 0;" : "=r"(borrow));

            asm("add.cc.u32 %0, %1, %2;" : "=r"(tmp[0]) : "r"(even[0]), "r"(MOD[0]));
            for (i = 1; i < n-1; i++)
                asm("addc.cc.u32 %0, %1, %2;" : "=r"(tmp[i]) : "r"(even[i]), "r"(MOD[i]));
            asm("addc.u32 %0, %1, %2;" : "=r"(tmp[i]) : "r"(even[i]), "r"(MOD[i]));

            asm("{ .reg.pred %top; setp.ne.u32 %top, %0, 0;" :: "r"(borrow));
            for (i = 0; i < n; i++)
                asm("@%top mov.b32 %0, %1;" : "+r"(even[i]) : "r"(tmp[i]));
            asm("}");

            return *this;
        }
        friend inline mont_t operator-(mont_t a, const mont_t& b)
        {   return a -= b;   }

        inline mont_t operator-() const
        {   return cneg(*this, true);   }

    private:
        static inline void madc_n_rshift(uint32_t* odd, const uint32_t *a, uint32_t bi)
        {
            for (size_t j = 0; j < n-2; j += 2)
                asm("madc.lo.cc.u32 %0, %2, %3, %4; madc.hi.cc.u32 %1, %2, %3, %5;"
                    : "=r"(odd[j]), "=r"(odd[j+1])
                    : "r"(a[j]), "r"(bi), "r"(odd[j+2]), "r"(odd[j+3]));
            asm("madc.lo.cc.u32 %0, %2, %3, 0; madc.hi.u32 %1, %2, %3, 0;"
                : "=r"(odd[n-2]), "=r"(odd[n-1])
                : "r"(a[n-2]), "r"(bi));
        }

        static inline void mad_n_redc(uint32_t *even, uint32_t* odd,
                                    const uint32_t *a, uint32_t bi, bool first=false)
        {
            if (first) {
                mul_n(odd, a+1, bi);
                mul_n(even, a,  bi);
            } else {
                asm("add.cc.u32 %0, %0, %1;" : "+r"(even[0]) : "r"(odd[1]));
                madc_n_rshift(odd, a+1, bi);
                cmad_n(even, a, bi);
                asm("addc.u32 %0, %0, 0;" : "+r"(odd[n-1]));
            }

            uint32_t mi = even[0] * M0;

            cmad_n(odd, MOD+1, mi);
            cmad_n(even, MOD,  mi);
            asm("addc.u32 %0, %0, 0;" : "+r"(odd[n-1]));
        }

    public:
        friend inline mont_t operator*(const mont_t& a, const mont_t& b)
        {
            if (N%32 == 0) {
                return wide_t{a, b};
            } else {
                mont_t even, odd;

                #pragma unroll
                for (size_t i = 0; i < n; i += 2) {
                    mad_n_redc(&even[0], &odd[0], &a[0], b[i], i==0);
                    mad_n_redc(&odd[0], &even[0], &a[0], b[i+1]);
                }

                // merge |even| and |odd|
                cadd_n(&even[0], &odd[1], n-1);
                asm("addc.u32 %0, %0, 0;" : "+r"(even[n-1]));

                even.final_sub(0, &odd[0]);

                return even;
            }
        }
        inline mont_t& operator*=(const mont_t& a)
        {   return *this = *this * a;   }

        inline mont_t& sqr()
        {   return *this = wide_t{*this};   }

        inline mont_t operator()(uint32_t p)
        {   return *this^p;   }

        // raise to a constant power, e.g. x^7, to be unrolled at compile time
        inline mont_t& operator^=(int p)
        {
            if (p < 2)
                asm("trap;");

            mont_t sqr = *this;
            if ((p&1) == 0) {
                do {
                    sqr.sqr();
                    p >>= 1;
                } while ((p&1) == 0);
                *this = sqr;
            }
            for (p >>= 1; p; p >>= 1) {
                sqr.sqr();
                if (p&1)
                    *this *= sqr;
            }
            return *this;
        }
        friend inline mont_t operator^(mont_t a, int p)
        {   return p == 2 ? (mont_t)wide_t{a} : a ^= p;   }
        inline mont_t operator()(int p)
        {   return *this^p;   }
        friend inline mont_t sqr(const mont_t& a)
        {   return a^2;   }

        inline void to()    { mont_t t = RR * *this; *this = t; }
        inline void to(const uint32_t a[2*n], bool host_order = true)
        {
            size_t i;

            // load the most significant half
            if (host_order) {
                for (i = 0; i < n; i++)
                    even[i] = a[n + i];
            } else {
                for (i = 0; i < n; i++)
                    asm("prmt.b32 %0, %1, %1, 0x0123;" : "=r"(even[i]) : "r"(a[n - 1 - i]));
            }
            to();

            mont_t lo;

            // load the least significant half
            if (host_order) {
                for (i = 0; i < n; i++)
                    lo[i] = a[i];
            } else {
                for (i = 0; i < n; i++)
                    asm("prmt.b32 %0, %1, %1, 0x0123;" : "=r"(lo[i]) : "r"(a[2*n - 1 - i]));
            }

            cadd_n(&even[0], &lo[0]);
            final_subc();
            to();
        }
        inline void from()  { mont_t t = *this; t.mul_by_1(); *this = t; }
        inline void from(const uint32_t a[2*n], bool host_order = true)
        {
            size_t i;

            // load the least significant half
            if (host_order) {
                for (i = 0; i < n; i++)
                    even[i] = a[i];
            } else {
                for (i = 0; i < n; i++)
                    asm("prmt.b32 %0, %1, 0, 0x0123;" : "=r"(even[i]) : "r"(a[2*n - 1 -i]));
            }
            mul_by_1();

            mont_t hi;

            // load the most significant half
            if (host_order) {
                for (i = 0; i < n; i++)
                    hi[i] = a[n + i];
            } else {
                for (i = 0; i < n; i++)
                    asm("prmt.b32 %0, %1, 0, 0x0123;" : "=r"(hi[i]) : "r"(a[n - 1 - i]));
            }

            cadd_n(&even[0], &hi[0]);
            final_subc();
            to();
        }

    public:
        static inline void mul_by_1_row(uint32_t* even, uint32_t* odd, bool first=false)
        {
            uint32_t mi;

            if (first) {
                mi = even[0] * M0;
                mul_n(odd, MOD+1, mi);
                cmad_n(even, MOD,  mi);
                asm("addc.u32 %0, %0, 0;" : "+r"(odd[n-1]));
            } else {
                asm("add.cc.u32 %0, %0, %1;" : "+r"(even[0]) : "r"(odd[1]));
    # if 1      // do we trust the compiler to *not* touch the carry flag here?
                mi = even[0] * M0;
    # else
                asm("mul.lo.u32 %0, %1, %2;" : "=r"(mi) : "r"(even[0]), "r"(M0));
    # endif
                madc_n_rshift(odd, MOD+1, mi);
                cmad_n(even, MOD, mi);
                asm("addc.u32 %0, %0, 0;" : "+r"(odd[n-1]));
            }
        }
        inline void mul_by_1()
        {
            mont_t odd;

            #pragma unroll
            for (size_t i = 0; i < n; i += 2) {
                mul_by_1_row(&even[0], &odd[0], i==0);
                mul_by_1_row(&odd[0], &even[0]);
            }

            cadd_n(&even[0], &odd[1], n-1);
            asm("addc.u32 %0, %0, 0;" : "+r"(even[n-1]));
        }

        inline void final_sub(uint32_t carry, uint32_t* tmp)
        {
            size_t i;
            asm("{ .reg.pred %top;");

            asm("sub.cc.u32 %0, %1, %2;" : "=r"(tmp[0]) : "r"(even[0]), "r"(MOD[0]));
            for (i = 1; i < n; i++)
                asm("subc.cc.u32 %0, %1, %2;" : "=r"(tmp[i]) : "r"(even[i]), "r"(MOD[i]));
            if (N%32 == 0)
                asm("subc.u32 %0, %0, 0; setp.eq.u32 %top, %0, 0;" : "+r"(carry));
            else
                asm("subc.u32 %0, 0, 0; setp.eq.u32 %top, %0, 0;" : "=r"(carry));

            for (i = 0; i < n; i++)
                asm("@%top mov.b32 %0, %1;" : "+r"(even[i]) : "r"(tmp[i]));

            asm("}");
        }

        inline void final_subc()
        {
            uint32_t carry, tmp[n];

            asm("addc.u32 %0, 0, 0;" : "=r"(carry));

            asm("sub.cc.u32 %0, %1, %2;" : "=r"(tmp[0]) : "r"(even[0]), "r"(MOD[0]));
            for (size_t i = 1; i < n; i++)
                asm("subc.cc.u32 %0, %1, %2;" : "=r"(tmp[i]) : "r"(even[i]), "r"(MOD[i]));
            asm("subc.u32 %0, %0, 0;" : "+r"(carry));

            asm("{ .reg.pred %top;");
            asm("setp.eq.u32 %top, %0, 0;" :: "r"(carry));
            for (size_t i = 0; i < n; i++)
                asm("@%top mov.b32 %0, %1;" : "+r"(even[i]) : "r"(tmp[i]));
            asm("}");
        }

};


typedef mont_t<254, device::ALT_BN128_P, device::ALT_BN128_M0,
                    device::ALT_BN128_RR, device::ALT_BN128_one,
                    device::ALT_BN128_Px4> fp_mont;


typedef mont_t<753, device::MNT4753_Fr_P, device::MNT4753_Fr_M0,
                    device::MNT4753_Fr_P, device::MNT4753_Fr_P,
                    device::MNT4753_Fr_P> MNT4753_Fr;

typedef mont_t<1023, device::Curve1024_P0, device::Curve1024_M0,
                    device::Curve1024_P0, device::Curve1024_P0,
                    device::Curve1024_P0> CURVE1024_Fr;

typedef mont_t<895, device::Curve896_P0, device::Curve896_M0,
                    device::Curve896_P0, device::Curve896_P0,
                    device::Curve896_P0> CURVE896_Fr;

void __global__ __launch_bounds__(1024) func1(uint32_t res[]){
    const uint32_t* p;
    const uint32_t* p1;
 uint32_t arr[28] =  {0x197958e9
    ,0x5b06dee7
    ,0x942faea1
    ,0xa0f76754
    ,0x2e80f0d5
    ,0x04748ffa
    ,0x892bf79e
    ,0xbb0b295c
    ,0x556edc31
    ,0x9921918c
    ,0xdda073ef
    ,0x9bda0b5f
    ,0x3ef9763f
    ,0x0ff103bc
    ,0x47a18384
    ,0x499b63de
    ,0x0e3e8f37
    ,0x97ff8158
    ,0x79e55fd3
    ,0x0848545f
    ,0x6ec08783
    ,0x32cb88e9
    ,0xb5106359
    ,0x90fffb20
    ,0x536e6db9
    ,0x1e5c9100
    ,0x7bcb2c8c
    ,0xfc13f9c8};

    // uint32_t arr[8] =  {0xd87cfd47,0x3c208c16,0x6871ca8d,0x97816a91,0x8181585d,0xb85045b6,0xe131a029,0x30644e72};//p,module
    // uint32_t arr[8] =  {0x11111111,0x11111111,0x11111111,0x11111111,0x11111111,0x11111111,0x11111111,0x11111111};
    p = arr;
    CURVE896_Fr exampleInstance(p);


    // uint32_t arr1[8] = {0x0,0x0,0x0,0x0,0x0,0x0,0x0,0x1};
    // R
        // TO_CUDA_T(0xd35d438dc58f0d9d), TO_CUDA_T(0x0a78eb28f5c70b3a),
        // TO_CUDA_T(0x666ea36f7879462c), TO_CUDA_T(0x0e0a77c19a07df2f)
    // uint32_t arr1[8] ={0x538afa89,0xf32cfc5b,0xd44501fb,0xb5e71911,0x0a417ff6,0x47ab1eff,0xcab8351f,0x06d89f71};//RR
 uint32_t arr1[28] ={0xccf8930a
    ,0x73700cc1
    ,0xb4d48631
    ,0x2c86d3de
    ,0x564d5aca
    ,0x25a1fa51
    ,0x5030d598
    ,0xca567213
    ,0x45935ee9
    ,0x22fbf7a8
    ,0x991ea02d
    ,0xf6efb34f
    ,0x16ec4993
    ,0x83d6900d
    ,0x1e76566a
    ,0xecef949e
    ,0x164c2dd1
    ,0xb9c23571
    ,0x32c2e9c0
    ,0x19f13b14
    ,0x49dc737c
    ,0xed52f842
    ,0xe3f820e5
    ,0xa0a71515
    ,0xdc2aaf93
    ,0x709275da
    ,0x2fd55779
    ,0xc1409417};
    p1 = arr1;
    CURVE896_Fr exampleInstance1(p1);
    exampleInstance*=exampleInstance1;
    for(int i=0;i<exampleInstance.n;i++){
        res[i]=exampleInstance[i];
    }
    // printf("\d",res[0]);
    res[0]=exampleInstance.bit_length();
     return;
 }

# undef inline
# undef asm



#include<iostream>
int main(){
  uint32_t res[64]={0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};


    uint32_t *res_d;
    hipMalloc((void**)&res_d,64*sizeof(uint32_t));
    hipMemcpy(res_d, res,64*sizeof(uint32_t), hipMemcpyHostToDevice);



int GRIDEDIM = 1;
while (true)
{
    printf("gridDIM = %d\n",GRIDEDIM);


int NUM_REPEATS = 10;
 float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        hipEventQuery(start);

        func1<<<GRIDEDIM,1024>>>(res_d);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, stop);
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    
    printf("Time = %g +- %g ms.\n", t_ave, t_err);

GRIDEDIM=GRIDEDIM*10;
if(GRIDEDIM==10000000){
    break;
}
}
    hipMemcpy(res,res_d,32*sizeof(uint32_t),hipMemcpyDeviceToHost);
    for(int i=0;i<32;i++){
        std::cout<<std::hex<<res[i]<<std::endl;
    }
    return 0;
}